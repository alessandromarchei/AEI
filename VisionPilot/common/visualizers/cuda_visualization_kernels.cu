#include "hip/hip_runtime.h"
#include "../include/cuda_visualization_kernels.hpp"

#ifdef CUDA_FOUND
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <vector>

namespace autoware_pov::common {



// CUDA kernel to create masks from tensors (for backend use)
__global__ void createMaskKernel(const float* input, unsigned char* output, int rows, int cols, int channels) {
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    int y = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= rows || y >= cols) return;

    int idx = x * cols + y;
    
    if (channels > 1) {
        // Multi-class: do argmax
        float max_score = -1e9f;
        int best_class = 0;
        
        for (int c = 0; c < channels; ++c) {
            // NCHW format: input[batch=0][channel=c][height=x][width=y]
            float score = input[c * rows * cols + x * cols + y];
            if (score > max_score) {
                max_score = score;
                best_class = c;
            }
        }
        
        // Convert class IDs for scene segmentation: Class 1 -> 255, others -> 0
        output[idx] = (best_class == 1) ? 255 : 0;
    } else {
        // Binary: threshold
        float value = input[idx];
        output[idx] = (value > 0.0f) ? 255 : 0;
    }
}

bool CudaVisualizationKernels::createMaskFromTensorCUDA(
    const float* tensor_data,
    const std::vector<int64_t>& tensor_shape,
    cv::Mat& output_mask
) {
    // Check CUDA availability
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess || deviceCount <= 0) {
        return false;
    }

    if (tensor_shape.size() != 4) {
        return false;
    }
    
    int rows = static_cast<int>(tensor_shape[2]);
    int cols = static_cast<int>(tensor_shape[3]); 
    int channels = static_cast<int>(tensor_shape[1]);
    
    size_t input_size = rows * cols * channels * sizeof(float);
    size_t output_size = rows * cols * sizeof(unsigned char);
    
    float* d_input;
    unsigned char* d_output;
    
    // Allocate GPU memory
    hipMalloc(&d_input, input_size);
    hipMalloc(&d_output, output_size);
    
    // Copy tensor to GPU
    hipMemcpy(d_input, tensor_data, input_size, hipMemcpyHostToDevice);
    
    // Launch kernel
    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
    
    createMaskKernel<<<grid, block>>>(d_input, d_output, rows, cols, channels);
    
    error = hipGetLastError();
    if (error != hipSuccess) {
        hipFree(d_input); 
        hipFree(d_output);
        return false;
    }
    
    // Copy result back
    output_mask = cv::Mat(rows, cols, CV_8UC1);
    hipMemcpy(output_mask.data, d_output, output_size, hipMemcpyDeviceToHost);
    
    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    
    return true;
}

// Direct tensor-to-colored visualization kernel (full pipeline)
__global__ void createColoredVisualizationKernel(
    const float* input, 
    uchar3* output, 
    int rows, 
    int cols, 
    int channels,
    int viz_type_id  // 0=scene, 1=domain
) {
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    int y = blockIdx.x * blockDim.x + threadIdx.x;

    if (x >= rows || y >= cols) return;

    int idx = x * cols + y;
    
    uchar3 color;
    
    if (channels > 1) {
        // Multi-class: do argmax
        float max_score = -1e9f;
        int best_class = 0;
        
        for (int c = 0; c < channels; ++c) {
            float score = input[c * rows * cols + x * cols + y];
            if (score > max_score) {
                max_score = score;
                best_class = c;
            }
        }
        
        // Color mapping based on viz type
        if (viz_type_id == 0) {  // Scene segmentation
            if (best_class == 1) {
                color = make_uchar3(0, 0, 255);  // BGR: Red for foreground
            } else {
                color = make_uchar3(0, 0, 0);    // BGR: Black for background
            }
        } else {  // Domain segmentation
            if (best_class == 0) {
                color = make_uchar3(61, 93, 255);   // BGR: Orange for road
            } else {
                color = make_uchar3(255, 28, 145);  // BGR: Purple for off-road
            }
        }
    } else {
        // Binary threshold
        float value = input[idx];
        if (value > 0.0f) {
            color = make_uchar3(0, 0, 255);  // BGR: Red
        } else {
            color = make_uchar3(0, 0, 0);    // BGR: Black
        }
    }

    output[idx] = color;
}

bool CudaVisualizationKernels::createColoredVisualizationCUDA(
    const float* tensor_data,
    const std::vector<int64_t>& tensor_shape,
    const std::string& viz_type,
    cv::Mat& colored_output
) {
    // Check CUDA availability
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess || deviceCount <= 0) {
        return false;
    }

    if (tensor_shape.size() != 4) {
        return false;
    }
    
    int rows = static_cast<int>(tensor_shape[2]);
    int cols = static_cast<int>(tensor_shape[3]); 
    int channels = static_cast<int>(tensor_shape[1]);
    
    size_t input_size = rows * cols * channels * sizeof(float);
    size_t output_size = rows * cols * sizeof(uchar3);
    
    float* d_input;
    uchar3* d_output;
    
    // Allocate GPU memory
    hipMalloc(&d_input, input_size);
    hipMalloc(&d_output, output_size);
    
    // Copy tensor to GPU
    hipMemcpy(d_input, tensor_data, input_size, hipMemcpyHostToDevice);
    
    // Launch kernel
    dim3 block(16, 16);
    dim3 grid((cols + block.x - 1) / block.x, (rows + block.y - 1) / block.y);
    
    int viz_type_id = (viz_type == "scene") ? 0 : 1;
    createColoredVisualizationKernel<<<grid, block>>>(d_input, d_output, rows, cols, channels, viz_type_id);
    
    error = hipGetLastError();
    if (error != hipSuccess) {
        hipFree(d_input); 
        hipFree(d_output);
        return false;
    }
    
    // Copy result back
    colored_output = cv::Mat(rows, cols, CV_8UC3);
    hipMemcpy(colored_output.data, d_output, output_size, hipMemcpyDeviceToHost);
    
    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    
    return true;
}

} // namespace autoware_pov::common

#endif // CUDA_FOUND